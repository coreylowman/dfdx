#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"

struct MinimumKernelOp {};

extern "C" __global__ void minimum_forward(
    const MinimumKernelOp op,
    const size_t numel,
    const size_t num_dims,
    const size_t *dims,
    const float *lhs,
    const size_t *lhs_strides,
    const float *rhs,
    const size_t *rhs_strides,
    float *out,
    const size_t *out_strides
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    unsigned int lhs_i = get_strided_index(i, num_dims, dims, lhs_strides);
    unsigned int rhs_i = get_strided_index(i, num_dims, dims, rhs_strides);
    unsigned int out_i = get_strided_index(i, num_dims, dims, out_strides);

    out[out_i] = fminf(lhs[lhs_i], rhs[rhs_i]);
}

extern "C" __global__ void minimum_backward(
    const MinimumKernelOp op,
    const size_t numel,
    const size_t num_dims,
    const size_t *dims,
    const float *lhs,
    float *grad_lhs,
    const size_t *lhs_strides,
    const float *rhs,
    float *grad_rhs,
    const size_t *rhs_strides,
    const float *grad_out,
    const size_t *out_strides
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    unsigned int lhs_i = get_strided_index(i, num_dims, dims, lhs_strides);
    unsigned int rhs_i = get_strided_index(i, num_dims, dims, rhs_strides);
    unsigned int out_i = get_strided_index(i, num_dims, dims, out_strides);

    auto x = lhs[lhs_i];
    auto y = rhs[rhs_i];
    auto go = grad_out[out_i];

    float dfdx, dfdy;

    if (x < y) {
        dfdx = 1.0;
        dfdy = 0.0;
    } else if (x > y) {
        dfdx = 0.0;
        dfdy = 1.0;
    } else {
        dfdx = 0.5;
        dfdy = 0.5;
    }

    atomicAdd(grad_lhs + lhs_i, dfdx * go);
    atomicAdd(grad_rhs + rhs_i, dfdy * go);
}
