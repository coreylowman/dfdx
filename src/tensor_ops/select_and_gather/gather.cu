#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"

__device__ unsigned int get_gathered_index(
    const unsigned int index,
    const size_t inp_num_dims,
    const size_t *inp_dims,
    const size_t *inp_strides,
    const size_t *idx,
    const size_t idx_num_dims,
    const size_t *idx_dims,
    const size_t *idx_strides,
    const size_t out_num_dims
) {
    unsigned int ax;

    if (out_num_dims > inp_num_dims) {
        ax = 0;
    } else {
        ax = idx_num_dims - 1;
    }

    unsigned int elem_size = 1; // the size of each indexed element
    unsigned int inp_row_len = inp_dims[ax]; // the size of the indexed dimension in the input
    unsigned int out_row_len = idx_dims[ax]; // the size of the indexed dimension in the output

    for (unsigned int d = 0; d < inp_num_dims - ax - 1; d++) {
        unsigned int dim_idx = inp_num_dims - 1 - d;
        elem_size *= inp_dims[dim_idx];
    }

    // location to find the index for the replaced dimension in "idx"
    unsigned int idx_idx = get_strided_index(index / elem_size, idx_num_dims, idx_dims, idx_strides);

    // indices for dimensions before, at, and after the indexed dimension
    unsigned int idx_before = index / (elem_size * out_row_len);
    unsigned int idx_mid = idx[idx_idx];
    assert(idx_mid < inp_dims[ax]);
    unsigned int idx_after = index % elem_size;

    // recombine
    unsigned int new_idx = (idx_before * inp_row_len + idx_mid) * elem_size + idx_after;
    return get_strided_index(new_idx, inp_num_dims, inp_dims, inp_strides);
}

template<typename T>
__device__ void gather_fwd(
    const size_t numel,
    const T *inp,
    const size_t inp_num_dims,
    const size_t *inp_dims,
    const size_t *inp_strides,
    const size_t *idx,
    const size_t idx_num_dims,
    const size_t *idx_dims,
    const size_t *idx_strides,
    T *out,
    const size_t out_num_dims
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    unsigned int out_i = i;
    unsigned int inp_i =
        get_gathered_index(i, inp_num_dims, inp_dims, inp_strides, idx, idx_num_dims, idx_dims, idx_strides, out_num_dims);

    out[out_i] = inp[inp_i];
    // out[out_i] = inp_i;
}

template<typename T>
__device__ void gather_bwd(
    const size_t numel,
    T *grad_inp,
    const size_t inp_num_dims,
    const size_t *inp_dims,
    const size_t *inp_strides,
    const size_t *idx,
    const size_t idx_num_dims,
    const size_t *idx_dims,
    const size_t *idx_strides,
    const T *grad_out,
    const size_t out_num_dims
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    unsigned int out_i = i;
    unsigned int inp_i =
        get_gathered_index(i, inp_num_dims, inp_dims, inp_strides, idx, idx_num_dims, idx_dims, idx_strides, out_num_dims);

    atomicAdd(grad_inp + inp_i, grad_out[out_i]);
}

#define GATHER(TYPENAME, FWD, BWD) \
extern "C" __global__ void FWD( \
    const size_t numel, \
    const TYPENAME *inp, \
    const size_t inp_num_dims, \
    const size_t *inp_dims, \
    const size_t *inp_strides, \
    const size_t *idx, \
    const size_t idx_num_dims, \
    const size_t *idx_dims, \
    const size_t *idx_strides, \
    TYPENAME *out, \
    const size_t out_num_dims \
) { \
    gather_fwd(numel, inp, inp_num_dims, inp_dims, inp_strides, idx, idx_num_dims, idx_dims, idx_strides, out, out_num_dims); \
} \
extern "C" __global__ void BWD( \
    const size_t numel, \
    TYPENAME *grad_inp, \
    const size_t inp_num_dims, \
    const size_t *inp_dims, \
    const size_t *inp_strides, \
    const size_t *idx, \
    const size_t idx_num_dims, \
    const size_t *idx_dims, \
    const size_t *idx_strides, \
    const TYPENAME *grad_out, \
    const size_t out_num_dims \
) { \
    gather_bwd(numel, grad_inp, inp_num_dims, inp_dims, inp_strides, idx, idx_num_dims, idx_dims, idx_strides, grad_out, out_num_dims); \
}

GATHER(__half, gather_fwd_f16, gather_bwd_f16);
GATHER(float, gather_fwd_f32, gather_bwd_f32);
GATHER(double, gather_fwd_f64, gather_bwd_f64);
