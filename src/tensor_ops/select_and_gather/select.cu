
#include <hip/hip_runtime.h>
__device__ unsigned int get_strided_index(
    unsigned int idx,
    const size_t num_dims,
    const size_t *dims,
    const size_t *strides
) {
    unsigned int strided_i = 0;
    for (unsigned int d = 0; d < num_dims; d++) {
        unsigned int dim_idx = num_dims - 1 - d;
        strided_i += (idx % dims[dim_idx]) * strides[dim_idx];
        idx /= dims[dim_idx];
    }
    return strided_i;
}

__device__ unsigned int get_selected_index(
    const unsigned int index,
    const size_t inp_num_dims,
    const size_t *inp_dims,
    const size_t *inp_strides,
    const size_t *idx,
    const size_t idx_num_dims,
    const size_t *idx_dims,
    const size_t *idx_strides
) {
    unsigned int elem_size = 1; // the size of each indexed element
    unsigned int row_len = inp_dims[idx_num_dims]; // the size of the indexed dimension

    for (unsigned int d = 0; d < inp_num_dims - idx_num_dims - 1; d++) {
        unsigned int dim_idx = inp_num_dims - 1 - d;
        elem_size *= inp_dims[dim_idx];
    }

    // indices for dimensions before, at, and after the indexed dimension
    unsigned int idx_before = index / elem_size;
    unsigned int idx_mid = idx[get_strided_index(idx_before, idx_num_dims, idx_dims, idx_strides)];
    unsigned int idx_after = index % elem_size;

    // recombine
    unsigned int new_idx = (idx_before * row_len + idx_mid) * elem_size + idx_after;
    return get_strided_index(new_idx, inp_num_dims, inp_dims, inp_strides);
}

extern "C" __global__ void select_forward(
    const size_t numel,
    const float *inp,
    const size_t inp_num_dims,
    const size_t *inp_dims,
    const size_t *inp_strides,
    const size_t *idx,
    const size_t idx_num_dims,
    const size_t *idx_dims,
    const size_t *idx_strides,
    float *out,
    const size_t *out_dims,
    const size_t *out_strides
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    unsigned int out_i = get_strided_index(i, inp_num_dims - 1, out_dims, out_strides);
    unsigned int inp_i =
        get_selected_index(i, inp_num_dims, inp_dims, inp_strides, idx, idx_num_dims, idx_dims, idx_strides);

    out[out_i] = inp[inp_i];
}

extern "C" __global__ void select_backward(
    const size_t numel,
    float *grad_inp,
    const size_t inp_num_dims,
    const size_t *inp_dims,
    const size_t *inp_strides,
    const size_t *idx,
    const size_t idx_num_dims,
    const size_t *idx_dims,
    const size_t *idx_strides,
    const float *grad_out,
    const size_t *out_dims,
    const size_t *out_strides
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    unsigned int out_i = get_strided_index(i, inp_num_dims - 1, out_dims, out_strides);
    unsigned int inp_i =
        get_selected_index(i, inp_num_dims, inp_dims, inp_strides, idx, idx_num_dims, idx_dims, idx_strides);

    atomicAdd(grad_inp + inp_i, grad_out[out_i]);
}
