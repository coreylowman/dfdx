#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"

__device__ unsigned int get_selected_index(
    const unsigned int index,
    const size_t inp_num_dims,
    const size_t *inp_dims,
    const size_t *inp_strides,
    const size_t *idx,
    const size_t idx_num_dims,
    const size_t *idx_dims,
    const size_t *idx_strides
) {
    unsigned int elem_size = 1; // the size of each indexed element
    unsigned int row_len = inp_dims[idx_num_dims]; // the size of the indexed dimension

    for (unsigned int d = 0; d < inp_num_dims - idx_num_dims - 1; d++) {
        unsigned int dim_idx = inp_num_dims - 1 - d;
        elem_size *= inp_dims[dim_idx];
    }

    // indices for dimensions before, at, and after the indexed dimension
    unsigned int idx_before = index / elem_size;
    unsigned int idx_mid = idx[get_strided_index(idx_before, idx_num_dims, idx_dims, idx_strides)];
    assert(idx_mid < inp_dims[idx_num_dims]);
    unsigned int idx_after = index % elem_size;

    // recombine
    unsigned int new_idx = (idx_before * row_len + idx_mid) * elem_size + idx_after;
    return get_strided_index(new_idx, inp_num_dims, inp_dims, inp_strides);
}

template<typename T>
__device__ void select_fwd(
    const size_t numel,
    const T *inp,
    const size_t inp_num_dims,
    const size_t *inp_dims,
    const size_t *inp_strides,
    const size_t *idx,
    const size_t idx_num_dims,
    const size_t *idx_dims,
    const size_t *idx_strides,
    T *out,
    const size_t *out_dims,
    const size_t *out_strides
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    unsigned int out_i = get_strided_index(i, inp_num_dims - 1, out_dims, out_strides);
    unsigned int inp_i =
        get_selected_index(i, inp_num_dims, inp_dims, inp_strides, idx, idx_num_dims, idx_dims, idx_strides);

    out[out_i] = inp[inp_i];
}

template<typename T>
__device__ void select_bwd(
    const size_t numel,
    T *grad_inp,
    const size_t inp_num_dims,
    const size_t *inp_dims,
    const size_t *inp_strides,
    const size_t *idx,
    const size_t idx_num_dims,
    const size_t *idx_dims,
    const size_t *idx_strides,
    const T *grad_out,
    const size_t *out_dims,
    const size_t *out_strides
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    unsigned int out_i = get_strided_index(i, inp_num_dims - 1, out_dims, out_strides);
    unsigned int inp_i =
        get_selected_index(i, inp_num_dims, inp_dims, inp_strides, idx, idx_num_dims, idx_dims, idx_strides);

    atomicAdd(grad_inp + inp_i, grad_out[out_i]);
}

#define SELECT(TYPENAME, FWD, BWD) \
extern "C" __global__ void FWD( \
    const size_t numel, \
    const TYPENAME *inp, \
    const size_t inp_num_dims, \
    const size_t *inp_dims, \
    const size_t *inp_strides, \
    const size_t *idx, \
    const size_t idx_num_dims, \
    const size_t *idx_dims, \
    const size_t *idx_strides, \
    TYPENAME *out, \
    const size_t *out_dims, \
    const size_t *out_strides \
) { \
    select_fwd(numel, inp, inp_num_dims, inp_dims, inp_strides, idx, idx_num_dims, idx_dims, idx_strides, out, out_dims, out_strides); \
} \
extern "C" __global__ void BWD( \
    const size_t numel, \
    TYPENAME *grad_inp, \
    const size_t inp_num_dims, \
    const size_t *inp_dims, \
    const size_t *inp_strides, \
    const size_t *idx, \
    const size_t idx_num_dims, \
    const size_t *idx_dims, \
    const size_t *idx_strides, \
    const TYPENAME *grad_out, \
    const size_t *out_dims, \
    const size_t *out_strides \
) { \
    select_bwd(numel, grad_inp, inp_num_dims, inp_dims, inp_strides, idx, idx_num_dims, idx_dims, idx_strides, grad_out, out_dims, out_strides); \
}

SELECT(float, select_fwd_f32, select_bwd_f32);
SELECT(double, select_fwd_f64, select_bwd_f64)
