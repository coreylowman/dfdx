#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"

__device__ unsigned int get_selected_index(
    const unsigned int index,
    const size_t inp_num_dims,
    const size_t *inp_dims,
    const size_t *inp_strides,
    const size_t *idx,
    const size_t idx_num_dims,
    const size_t *idx_dims,
    const size_t *idx_strides
) {
    unsigned int elem_size = 1; // the size of each indexed element
    unsigned int row_len = inp_dims[idx_num_dims]; // the size of the indexed dimension

    for (unsigned int d = 0; d < inp_num_dims - idx_num_dims - 1; d++) {
        unsigned int dim_idx = inp_num_dims - 1 - d;
        elem_size *= inp_dims[dim_idx];
    }

    // indices for dimensions before, at, and after the indexed dimension
    unsigned int idx_before = index / elem_size;
    unsigned int idx_mid = idx[get_strided_index(idx_before, idx_num_dims, idx_dims, idx_strides)];
    unsigned int idx_after = index % elem_size;

    // recombine
    unsigned int new_idx = (idx_before * row_len + idx_mid) * elem_size + idx_after;
    return get_strided_index(new_idx, inp_num_dims, inp_dims, inp_strides);
}

template<typename T>
__device__ void select_forward(
    const size_t numel,
    const T *inp,
    const size_t inp_num_dims,
    const size_t *inp_dims,
    const size_t *inp_strides,
    const size_t *idx,
    const size_t idx_num_dims,
    const size_t *idx_dims,
    const size_t *idx_strides,
    T *out,
    const size_t *out_dims,
    const size_t *out_strides
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    unsigned int out_i = get_strided_index(i, inp_num_dims - 1, out_dims, out_strides);
    unsigned int inp_i =
        get_selected_index(i, inp_num_dims, inp_dims, inp_strides, idx, idx_num_dims, idx_dims, idx_strides);

    out[out_i] = inp[inp_i];
}

template<typename T>
__device__ void select_backward(
    const size_t numel,
    T *grad_inp,
    const size_t inp_num_dims,
    const size_t *inp_dims,
    const size_t *inp_strides,
    const size_t *idx,
    const size_t idx_num_dims,
    const size_t *idx_dims,
    const size_t *idx_strides,
    const T *grad_out,
    const size_t *out_dims,
    const size_t *out_strides
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    unsigned int out_i = get_strided_index(i, inp_num_dims - 1, out_dims, out_strides);
    unsigned int inp_i =
        get_selected_index(i, inp_num_dims, inp_dims, inp_strides, idx, idx_num_dims, idx_dims, idx_strides);

    atomicAdd(grad_inp + inp_i, grad_out[out_i]);
}

extern "C" __global__ void select_forward_f32(
    const size_t numel,
    const float *inp,
    const size_t inp_num_dims,
    const size_t *inp_dims,
    const size_t *inp_strides,
    const size_t *idx,
    const size_t idx_num_dims,
    const size_t *idx_dims,
    const size_t *idx_strides,
    float *out,
    const size_t *out_dims,
    const size_t *out_strides
) {
    select_forward(numel, inp, inp_num_dims, inp_dims, inp_strides, idx, idx_num_dims, idx_dims, idx_strides, out, out_dims, out_strides);
}

extern "C" __global__ void select_backward_f32(
    const size_t numel,
    float *grad_inp,
    const size_t inp_num_dims,
    const size_t *inp_dims,
    const size_t *inp_strides,
    const size_t *idx,
    const size_t idx_num_dims,
    const size_t *idx_dims,
    const size_t *idx_strides,
    const float *grad_out,
    const size_t *out_dims,
    const size_t *out_strides
) {
    select_backward(numel, grad_inp, inp_num_dims, inp_dims, inp_strides, idx, idx_num_dims, idx_dims, idx_strides, grad_out, out_dims, out_strides);
}

extern "C" __global__ void select_forward_f64(
    const size_t numel,
    const double *inp,
    const size_t inp_num_dims,
    const size_t *inp_dims,
    const size_t *inp_strides,
    const size_t *idx,
    const size_t idx_num_dims,
    const size_t *idx_dims,
    const size_t *idx_strides,
    double *out,
    const size_t *out_dims,
    const size_t *out_strides
) {
    select_forward(numel, inp, inp_num_dims, inp_dims, inp_strides, idx, idx_num_dims, idx_dims, idx_strides, out, out_dims, out_strides);
}

extern "C" __global__ void select_backward_f64(
    const size_t numel,
    double *grad_inp,
    const size_t inp_num_dims,
    const size_t *inp_dims,
    const size_t *inp_strides,
    const size_t *idx,
    const size_t idx_num_dims,
    const size_t *idx_dims,
    const size_t *idx_strides,
    const double *grad_out,
    const size_t *out_dims,
    const size_t *out_strides
) {
    select_backward(numel, grad_inp, inp_num_dims, inp_dims, inp_strides, idx, idx_num_dims, idx_dims, idx_strides, grad_out, out_dims, out_strides);
}