#include "hip/hip_runtime.h"
#include "unary_op_macros.cuh"

template<typename F>
struct PowFKernelOp {
    F rhs;
};

UNARY_OP(float, pow_fwd_f32, pow_bwd_f32, PowFKernelOp<float>,
        powf(x, op.rhs),
        op.rhs * powf(x, op.rhs - 1.0))

UNARY_OP(double, pow_fwd_f64, pow_bwd_f64, PowFKernelOp<double>,
    pow(x, op.rhs),
    op.rhs * pow(x, op.rhs - 1.0))
    