#include "hip/hip_runtime.h"
#include "binary_op_macros.cuh"

struct BCEKernelOp {};

LONG_BINARY_OP(bce_forward, bce_backward, BCEKernelOp,
    {
        float logit = lhs[lhs_i];
        float prob = rhs[rhs_i];

        fx = fmaxf(logit, 0.0) - logit * prob + logf(1.0 + expf(-fabsf(logit)));
    },
    {
        auto logit = lhs[lhs_i];
        auto prob = rhs[rhs_i];
        auto go = grad_out[out_i];

        dfdx = 1.0 - prob - 1 / (1.0 + expf(logit));
        dfdy = -logit;
    }
)
