#include "hip/hip_runtime.h"
#include "unary_op_macros.cuh"

struct SqrtKernelOp {};

UNARY_OP(float, sqrt_fwd_f32, sqrt_bwd_f32, SqrtKernelOp,
        sqrtf(x),
        1 / (y + y))

UNARY_OP(double, sqrt_fwd_f64, sqrt_bwd_f64, SqrtKernelOp,
        sqrt(x),
        1 / (y + y))
        