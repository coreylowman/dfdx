#include "hip/hip_runtime.h"
#include "unary_op_macros.cuh"

struct SqrtKernelOp {};

UNARY_OP(__half, sqrt_fwd_f16, sqrt_bwd_f16, SqrtKernelOp,
        sqrtg(x),
        recipg(y + y))

UNARY_OP(float, sqrt_fwd_f32, sqrt_bwd_f32, SqrtKernelOp,
        sqrtg(x),
        recipg(y + y))

UNARY_OP(double, sqrt_fwd_f64, sqrt_bwd_f64, SqrtKernelOp,
        sqrtg(x),
        recipg(y + y))
        