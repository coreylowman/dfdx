
#include <hip/hip_runtime.h>
#include "hip/hip_fp16.h"

#define DROPOUT(TYPENAME, FWD, BWD) \
extern "C" __global__ void FWD( \
    const TYPENAME prob, \
    const size_t numel, \
    const TYPENAME *inp, \
    const TYPENAME *noise, \
    TYPENAME *out \
) { \
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; \
    if (i >= numel) { \
        return; \
    } \
    TYPENAME zero = 0.0; \
    TYPENAME one = 1.0; \
    TYPENAME scalar = (noise[i] < prob) ? zero : (one / (one - prob)); \
    out[i] = inp[i] * scalar; \
} \
extern "C" __global__ void BWD( \
    const TYPENAME prob, \
    const size_t numel, \
    const TYPENAME *noise, \
    TYPENAME *grad_inp, \
    const TYPENAME *grad_out \
) { \
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; \
    if (i >= numel) { \
        return; \
    } \
    TYPENAME zero = 0.0; \
    TYPENAME one = 1.0; \
    grad_inp[i] += (noise[i] < prob) ? zero : (grad_out[i] / (one - prob)); \
}

DROPOUT(__half, dropout_fwd_f16, dropout_bwd_f16);
DROPOUT(float, dropout_fwd_f32, dropout_bwd_f32);
DROPOUT(double, dropout_fwd_f64, dropout_bwd_f64);
