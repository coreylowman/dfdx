
#include <hip/hip_runtime.h>
struct BinaryAddOp {};

extern "C" __global__ void binary_add_forward(
    const BinaryAddOp op,
    const size_t numel,
    const float *lhs,
    const float *rhs,
    float *out
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    out[i] = lhs[i] + rhs[i];
}

extern "C" __global__ void binary_add_backward(
    const BinaryAddOp op,
    const size_t numel,
    const float *lhs,
    float *grad_lhs,
    const float *rhs,
    float *grad_rhs,
    const float *grad_out
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    auto x = lhs[i];
    auto y = rhs[i];
    auto go = grad_out[i];

    grad_lhs[i] += go;
    grad_rhs[i] += go;
}
