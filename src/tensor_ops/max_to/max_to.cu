#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"

__device__ __forceinline__ __half atomicMaxf(__half* address, __half val) {
#if __CUDA_ARCH__ < 700
    // On older GPUs we do not have access to atomicCAS for shorts, so we have to do some trickery.
    // Solution adapted from https://github.com/torch/cutorch/blob/master/lib/THC/THCAtomics.cuh#L96-L119
    unsigned int *address_as_ui = (unsigned int *) ((char *)address - ((size_t)address & 2));
    unsigned int old = *address_as_ui;
    unsigned int assumed;
    bool unaligned = (size_t) address & 2;
    do {
        assumed = old;
        unsigned int hmax;
        hmax = unaligned ? (old >> 16) : (old & 0xffff);
        hmax = __half_as_ushort(__hmax_nan(val, __ushort_as_half(hmax))); 
        old = atomicCAS(address_as_ui, assumed,
            unaligned ? (old & 0xffff) | (hmax << 16) : (old & 0xffff0000) | hmax
        );

   } while (assumed != old);
   return __ushort_as_half(unaligned ? (old >> 16) : (old & 0xffff));
#else
    // Based on https://docs.nvidia.com/cuda/cuda-c-programming-guide/#atomic-functions
    unsigned short int* casted_address = (unsigned short int*)address;
    unsigned short int old = *casted_address;
    unsigned short int assumed;
    do {
        assumed = old;
        old = atomicCAS(casted_address, assumed, __half_as_ushort(__hmax_nan(val, __ushort_as_half(assumed))));
    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);
    return __ushort_as_half(old);
#endif
}

// atomicMax is not implemented for floats,
// solution copied https://stackoverflow.com/questions/17399119/how-do-i-use-atomicmax-on-floating-point-values-in-cuda
__device__ __forceinline__ float atomicMaxf(float * addr, float value) {
    if (signbit(value)) {
        return __uint_as_float(atomicMin((unsigned int *)addr, __float_as_uint(value)));        
    } else {
        return __int_as_float(atomicMax((int *)addr, __float_as_int(value)));
    }
}

__device__ __forceinline__ double atomicMaxf(double * addr, double value) {
    if (signbit(value)) {
        return __longlong_as_double(atomicMin((unsigned long long int *)addr, __double_as_longlong(value)));
    } else {
        return __longlong_as_double(atomicMax((long long int *)addr, __double_as_longlong(value)));
    }
}

// Efficiently computes the max of each chunk in "data" of size chunk_len, and
// stores the maximums in out[i / chunk_len]
template<typename T>
__device__ void chunk_max(
    const size_t numel,
    const size_t chunk_len,
    const T data,
    T* out
) {
    __shared__ T buf[1024];
    // assumes that threads where i >= numel have already exited
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int block_i = threadIdx.x;
    buf[block_i] = data;

    unsigned int chunk_i = i % chunk_len;
    unsigned int chunk_start = max((int)(block_i - chunk_i), 0);
    unsigned int chunk_end = min((unsigned int)(block_i + chunk_len - chunk_i), blockDim.x);

    chunk_i = block_i - chunk_start;

    size_t max_chunk_len = min(chunk_end - chunk_start, blockDim.x);
    size_t incr = next_power_of_two(max_chunk_len) >> 1;

    __syncthreads();

    // Uses sequential addressing as discussed in
    // https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
    for (; incr > 0; incr >>= 1) {
        unsigned int block_i_2 = block_i + incr;

        if (block_i_2 < chunk_end && chunk_i < incr) {
            // This is sound because __syncthreads and the conditions above
            // ensure that no data races occur
            buf[block_i] = maxg(buf[block_i], buf[block_i_2]);
        }

        __syncthreads();
    }

    if (block_i == chunk_start) {
        atomicMaxf(out + i / chunk_len, buf[block_i]);
    }
}

// strides and dims specify how to index inp to put all summed elements next to
// each other, and chunk_len is len(inp) / len(out)
template<typename T>
__device__ void max_to_fwd(
    const size_t numel,
    const size_t num_dims,
    const size_t chunk_len,
    const size_t *info,
    const T *inp,
    T *out
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= numel) {
        return;
    }

    const size_t *dims = info;
    const size_t *strides = info + num_dims;

    unsigned int inp_i = get_strided_index(i, num_dims, dims, strides);
    chunk_max(numel, chunk_len, inp[inp_i], out);
}

// Accepts pre-broadcasted strides for both input & output.
// So both inp & out are expected to be broadcasted to the same size.
template<typename T>
__device__ void max_to_bwd(
    const size_t numel,
    const size_t num_dims,
    const T elems_per_thread,
    const size_t *info,
    const T *inp,
    T *grad_inp,
    const T *out,
    const T *grad_out
) {
    unsigned int inp_i = blockIdx.x * blockDim.x + threadIdx.x;

    if (inp_i >= numel) {
        return;
    }

    const size_t *dims = info;
    const size_t *inp_strides = info + num_dims;
    const size_t *out_strides = info + 2 * num_dims;

    unsigned int out_i = restrided(inp_i, num_dims, dims, inp_strides, out_strides);

    const T mask = static_cast<T>(inp[inp_i] == out[out_i]);
    grad_inp[inp_i] += mask * grad_out[out_i] * elems_per_thread;
}

#define MAX(TYPENAME, FWD, BWD) \
extern "C" __global__ void FWD( \
    const size_t numel, \
    const size_t num_dims, \
    const size_t chunk_len, \
    const size_t *info, \
    const TYPENAME *inp, \
    TYPENAME *out \
) { \
    max_to_fwd(numel, num_dims, chunk_len, info, inp, out); \
} \
extern "C" __global__ void BWD( \
    const size_t numel, \
    const size_t num_dims, \
    const TYPENAME elems_per_thread, \
    const size_t *info, \
    const TYPENAME *inp, \
    TYPENAME *grad_inp, \
    const TYPENAME *out, \
    const TYPENAME *grad_out \
) { \
    max_to_bwd(numel, num_dims, elems_per_thread, info, inp, grad_inp, out, grad_out); \
}

MAX(__half, max_to_fwd_f16, max_to_bwd_f16);
MAX(float, max_to_fwd_f32, max_to_bwd_f32);
MAX(double, max_to_fwd_f64, max_to_bwd_f64);
