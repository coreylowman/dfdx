
#include <hip/hip_runtime.h>
__device__ unsigned int get_strided_index(
    unsigned int idx,
    const size_t num_dims,
    const size_t *dims,
    const size_t *strides
) {
    unsigned int strided_i = 0;
    for (unsigned int d = 0; d < num_dims; d++) {
        unsigned int dim_idx = num_dims - 1 - d;
        strided_i += (idx % dims[dim_idx]) * strides[dim_idx];
        idx /= dims[dim_idx];
    }
    return strided_i;
}

extern "C" __global__ void reshape_forward(
    const size_t numel,
    const float *inp,
    const size_t inp_num_dims,
    const size_t *inp_dims,
    const size_t *inp_strides,
    float *out,
    const size_t out_num_dims,
    const size_t *out_dims,
    const size_t *out_strides
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    unsigned int inp_i = get_strided_index(i, inp_num_dims, inp_dims, inp_strides);
    unsigned int out_i = get_strided_index(i, out_num_dims, out_dims, out_strides);

    out[out_i] = inp[inp_i];
}

extern "C" __global__ void reshape_backward(
    const size_t numel,
    float *grad_inp,
    const size_t inp_num_dims,
    const size_t *inp_dims,
    const size_t *inp_strides,
    const float *grad_out,
    const size_t out_num_dims,
    const size_t *out_dims,
    const size_t *out_strides
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    unsigned int inp_i = get_strided_index(i, inp_num_dims, inp_dims, inp_strides);
    unsigned int out_i = get_strided_index(i, out_num_dims, out_dims, out_strides);

    atomicAdd(grad_inp + inp_i, grad_out[out_i]);
}
