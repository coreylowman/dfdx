#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"

struct Pool2dOp {
    size_t kernel;
    size_t stride;
    size_t padding;
    size_t batch;
    size_t chan;
    size_t h_in;
    size_t h_out;
    size_t w_in;
    size_t w_out;
};

template<typename T>
__device__ void avg_pool2d_fwd(
    const Pool2dOp op,
    const size_t *inp_strides,
    const size_t *out_strides,
    const T *inp, // 4d (Batch, Channels, Height, Width)
    T *out // 4d (Batch, Channels, HeightOut, WidthOut)
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t numel = op.batch * op.chan * op.h_out * op.w_out;
    if (i >= numel) {
        return;
    }

    unsigned int idx = i;
    const size_t ow = idx % op.w_out;
    idx /= op.w_out;
    const size_t oh = idx % op.h_out;
    idx /= op.h_out;
    const size_t c = idx % op.chan;
    idx /= op.chan;
    const size_t b = idx % op.batch;
    idx /= op.batch;
    
    T tmp = 0.0;
    for(size_t k1 = 0; k1 < op.kernel; k1++) {
        for (size_t k2 = 0; k2 < op.kernel; k2++) {
            const size_t y_plus_p = oh * op.stride + k1;
            if (y_plus_p < op.padding) { continue; }
            const size_t y = y_plus_p - op.padding;
            if (y >= op.h_in) { continue; }
            const size_t x_plus_p = ow * op.stride + k2;
            if (x_plus_p < op.padding) { continue; }
            const size_t x = x_plus_p - op.padding;
            if (x >= op.w_in) { continue; }

            auto inp_i = b * inp_strides[0] + c * inp_strides[1] + y * inp_strides[2] + x * inp_strides[3];
            tmp += inp[inp_i];
        }
    }

    tmp /= static_cast<T>(op.kernel * op.kernel);
    out[i] = tmp;
}

template<typename T>
__device__ void avg_pool2d_bwd(
    const Pool2dOp op,
    const size_t *inp_strides,
    const size_t *out_strides,
    const T *inp, // 4d (Batch, Channels, Height, Width)
    T *grad_inp,
    const T *out, // 4d (Batch, Channels, HeightOut, WidthOut)
    const T *grad_out
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t numel = op.batch * op.chan * op.h_in * op.w_in;
    if (i >= numel) {
        return;
    }

    unsigned int idx = i;
    const size_t x = idx % op.w_in;
    idx /= op.w_in;
    const size_t y = idx % op.h_in;
    idx /= op.h_in;
    const size_t c = idx % op.chan;
    idx /= op.chan;
    const size_t b = idx % op.batch;
    idx /= op.batch;

    T tmp = 0.0;
    for(size_t k1 = 0; k1 < op.kernel; k1++) {
        for (size_t k2 = 0; k2 < op.kernel; k2++) {
            size_t oh = y + op.padding;
            if (oh < k1) { continue; }
            oh -= k1;
            if (oh % op.stride != 0) { continue; }
            oh /= op.stride;
            if (oh >= op.h_out) { continue; }

            size_t ow = x + op.padding;
            if (ow < k2) { continue; }
            ow -= k2;
            if (ow % op.stride != 0) { continue; }
            ow /= op.stride;
            if (ow >= op.w_out) { continue; }

            auto out_i = b * out_strides[0] + c * out_strides[1] + oh * out_strides[2] + ow * out_strides[3];
            tmp += grad_out[out_i];
        }
    }

    grad_inp[i] += tmp / static_cast<T>(op.kernel * op.kernel);
}

template<typename T>
__device__ void max_pool2d_fwd(
    const Pool2dOp op,
    const size_t *inp_strides,
    const size_t *out_strides,
    const T *inp, // 4d (Batch, Channels, Height, Width)
    T *out // 4d (Batch, Channels, HeightOut, WidthOut)
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t numel = op.batch * op.chan * op.h_out * op.w_out;
    if (i >= numel) {
        return;
    }

    unsigned int idx = i;
    const size_t ow = idx % op.w_out;
    idx /= op.w_out;
    const size_t oh = idx % op.h_out;
    idx /= op.h_out;
    const size_t c = idx % op.chan;
    idx /= op.chan;
    const size_t b = idx % op.batch;
    idx /= op.batch;

    T tmp = -INFINITY;
    for(size_t k1 = 0; k1 < op.kernel; k1++) {
        for (size_t k2 = 0; k2 < op.kernel; k2++) {
            const size_t y_plus_p = oh * op.stride + k1;
            if (y_plus_p < op.padding) { continue; }
            const size_t y = y_plus_p - op.padding;
            if (y >= op.h_in) { continue; }
            const size_t x_plus_p = ow * op.stride + k2;
            if (x_plus_p < op.padding) { continue; }
            const size_t x = x_plus_p - op.padding;
            if (x >= op.w_in) { continue; }

            auto inp_i = b * inp_strides[0] + c * inp_strides[1] + y * inp_strides[2] + x * inp_strides[3];
            tmp = maxg(tmp, inp[inp_i]);
        }
    }

    out[i] = tmp;
}

template<typename T>
__device__ void max_pool2d_bwd(
    const Pool2dOp op,
    const size_t *inp_strides,
    const size_t *out_strides,
    const T *inp, // 4d (Batch, Channels, Height, Width)
    T *grad_inp,
    const T *out, // 4d (Batch, Channels, HeightOut, WidthOut)
    const T *grad_out
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t numel = op.batch * op.chan * op.h_in * op.w_in;
    if (i >= numel) {
        return;
    }

    unsigned int idx = i;
    const size_t x = idx % op.w_in;
    idx /= op.w_in;
    const size_t y = idx % op.h_in;
    idx /= op.h_in;
    const size_t c = idx % op.chan;
    idx /= op.chan;
    const size_t b = idx % op.batch;
    idx /= op.batch;

    const T inp_v = inp[i];

    T tmp = 0.0;
    for(size_t k1 = 0; k1 < op.kernel; k1++) {
        for (size_t k2 = 0; k2 < op.kernel; k2++) {
            size_t oh = y + op.padding;
            if (oh < k1) { continue; }
            oh -= k1;
            if (oh % op.stride != 0) { continue; }
            oh /= op.stride;
            if (oh >= op.h_out) { continue; }

            size_t ow = x + op.padding;
            if (ow < k2) { continue; }
            ow -= k2;
            if (ow % op.stride != 0) { continue; }
            ow /= op.stride;
            if (ow >= op.w_out) { continue; }

            auto out_i = b * out_strides[0] + c * out_strides[1] + oh * out_strides[2] + ow * out_strides[3];

            if (out[out_i] == inp_v) {
                tmp += grad_out[out_i];
            }
        }
    }

    grad_inp[i] += tmp;
}

template<typename T>
__device__ void min_pool2d_fwd(
    const Pool2dOp op,
    const size_t *inp_strides,
    const size_t *out_strides,
    const T *inp, // 4d (Batch, Channels, Height, Width)
    T *out // 4d (Batch, Channels, HeightOut, WidthOut)
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t numel = op.batch * op.chan * op.h_out * op.w_out;
    if (i >= numel) {
        return;
    }

    unsigned int idx = i;
    const size_t ow = idx % op.w_out;
    idx /= op.w_out;
    const size_t oh = idx % op.h_out;
    idx /= op.h_out;
    const size_t c = idx % op.chan;
    idx /= op.chan;
    const size_t b = idx % op.batch;
    idx /= op.batch;

    T tmp = INFINITY;
    for(size_t k1 = 0; k1 < op.kernel; k1++) {
        for (size_t k2 = 0; k2 < op.kernel; k2++) {
            const size_t y_plus_p = oh * op.stride + k1;
            if (y_plus_p < op.padding) { continue; }
            const size_t y = y_plus_p - op.padding;
            if (y >= op.h_in) { continue; }
            const size_t x_plus_p = ow * op.stride + k2;
            if (x_plus_p < op.padding) { continue; }
            const size_t x = x_plus_p - op.padding;
            if (x >= op.w_in) { continue; }

            auto inp_i = b * inp_strides[0] + c * inp_strides[1] + y * inp_strides[2] + x * inp_strides[3];
            tmp = ming(tmp, inp[inp_i]);
        }
    }

    out[i] = tmp;
}

template<typename T>
__device__ void min_pool2d_bwd(
    const Pool2dOp op,
    const size_t *inp_strides,
    const size_t *out_strides,
    const T *inp, // 4d (Batch, Channels, Height, Width)
    T *grad_inp,
    const T *out, // 4d (Batch, Channels, HeightOut, WidthOut)
    const T *grad_out
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t numel = op.batch * op.chan * op.h_in * op.w_in;
    if (i >= numel) {
        return;
    }

    unsigned int idx = i;
    const size_t x = idx % op.w_in;
    idx /= op.w_in;
    const size_t y = idx % op.h_in;
    idx /= op.h_in;
    const size_t c = idx % op.chan;
    idx /= op.chan;
    const size_t b = idx % op.batch;
    idx /= op.batch;

    const T inp_v = inp[i];

    T tmp = 0.0;
    for(size_t k1 = 0; k1 < op.kernel; k1++) {
        for (size_t k2 = 0; k2 < op.kernel; k2++) {
            size_t oh = y + op.padding;
            if (oh < k1) { continue; }
            oh -= k1;
            if (oh % op.stride != 0) { continue; }
            oh /= op.stride;
            if (oh >= op.h_out) { continue; }

            size_t ow = x + op.padding;
            if (ow < k2) { continue; }
            ow -= k2;
            if (ow % op.stride != 0) { continue; }
            ow /= op.stride;
            if (ow >= op.w_out) { continue; }

            auto out_i = b * out_strides[0] + c * out_strides[1] + oh * out_strides[2] + ow * out_strides[3];

            if (out[out_i] == inp_v) {
                tmp += grad_out[out_i];
            }
        }
    }

    grad_inp[i] += tmp;
}

#define POOL_OP(TYPENAME, fwd, bwd, fwd_FN, bwd_FN) \
extern "C" __global__ void fwd( \
    const Pool2dOp op, \
    const size_t *inp_strides, \
    const size_t *out_strides, \
    const TYPENAME *inp, \
    TYPENAME *out \
) { \
    fwd_FN(op, inp_strides, out_strides, inp, out); \
} \
extern "C" __global__ void bwd( \
    const Pool2dOp op, \
    const size_t *inp_strides, \
    const size_t *out_strides, \
    const TYPENAME *inp, \
    TYPENAME *grad_inp, \
    const TYPENAME *out, \
    const TYPENAME *grad_out \
) { \
    bwd_FN(op, inp_strides, out_strides, inp, grad_inp, out, grad_out); \
}

POOL_OP(
    float,
    avg_pool2d_fwd_f32, avg_pool2d_bwd_f32,
    avg_pool2d_fwd, avg_pool2d_bwd
);
POOL_OP(
    float,
    min_pool2d_fwd_f32, min_pool2d_bwd_f32,
    min_pool2d_fwd, min_pool2d_bwd
);
POOL_OP(
    float,
    max_pool2d_fwd_f32, max_pool2d_bwd_f32,
    max_pool2d_fwd, max_pool2d_bwd
);

POOL_OP(
    double,
    avg_pool2d_fwd_f64, avg_pool2d_bwd_f64,
    avg_pool2d_fwd, avg_pool2d_bwd
);
POOL_OP(
    double,
    min_pool2d_fwd_f64, min_pool2d_bwd_f64,
    min_pool2d_fwd, min_pool2d_bwd
);
POOL_OP(
    double,
    max_pool2d_fwd_f64, max_pool2d_bwd_f64,
    max_pool2d_fwd, max_pool2d_bwd
);
