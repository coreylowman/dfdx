
#include <hip/hip_runtime.h>
struct Pool2dOp {
    size_t kernel;
    size_t stride;
    size_t padding;
    size_t batch;
    size_t chan;
    size_t h_in;
    size_t h_out;
    size_t w_in;
    size_t w_out;
};

extern "C" __global__ void avg_pool2d_forward(
    const Pool2dOp op,
    const size_t *inp_strides,
    const size_t *out_strides,
    const float *inp, // 4d (Batch, Channels, Height, Width)
    float *out // 4d (Batch, Channels, HeightOut, WidthOut)
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t numel = op.batch * op.chan * op.h_out * op.w_out;
    if (i >= numel) {
        return;
    }

    unsigned int idx = i;
    const size_t ow = idx % op.w_out;
    idx /= op.w_out;
    const size_t oh = idx % op.h_out;
    idx /= op.h_out;
    const size_t c = idx % op.chan;
    idx /= op.chan;
    const size_t b = idx % op.batch;
    idx /= op.batch;
    
    float tmp = 0.0;
    for(size_t k1 = 0; k1 < op.kernel; k1++) {
        for (size_t k2 = 0; k2 < op.kernel; k2++) {
            const size_t y_plus_p = oh * op.stride + k1;
            if (y_plus_p < op.padding) { continue; }
            const size_t y = y_plus_p - op.padding;
            if (y >= op.h_in) { continue; }
            const size_t x_plus_p = ow * op.stride + k2;
            if (x_plus_p < op.padding) { continue; }
            const size_t x = x_plus_p - op.padding;
            if (x >= op.w_in) { continue; }

            auto inp_i = b * inp_strides[0] + c * inp_strides[1] + y * inp_strides[2] + x * inp_strides[3];
            tmp += inp[inp_i];
        }
    }

    tmp /= static_cast<float>(op.kernel * op.kernel);
    out[i] = tmp;
}

extern "C" __global__ void avg_pool2d_backward(
    const Pool2dOp op,
    const size_t *inp_strides,
    const size_t *out_strides,
    const float *inp, // 4d (Batch, Channels, Height, Width)
    float *grad_inp,
    const float *out, // 4d (Batch, Channels, HeightOut, WidthOut)
    const float *grad_out
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t numel = op.batch * op.chan * op.h_in * op.w_in;
    if (i >= numel) {
        return;
    }

    unsigned int idx = i;
    const size_t x = idx % op.w_in;
    idx /= op.w_in;
    const size_t y = idx % op.h_in;
    idx /= op.h_in;
    const size_t c = idx % op.chan;
    idx /= op.chan;
    const size_t b = idx % op.batch;
    idx /= op.batch;

    float tmp = 0.0;
    for(size_t k1 = 0; k1 < op.kernel; k1++) {
        for (size_t k2 = 0; k2 < op.kernel; k2++) {
            size_t oh = y + op.padding;
            if (oh < k1) { continue; }
            oh -= k1;
            if (oh % op.stride != 0) { continue; }
            oh /= op.stride;
            if (oh >= op.h_out) { continue; }

            size_t ow = x + op.padding;
            if (ow < k2) { continue; }
            ow -= k2;
            if (ow % op.stride != 0) { continue; }
            ow /= op.stride;
            if (ow >= op.w_out) { continue; }

            auto out_i = b * out_strides[0] + c * out_strides[1] + oh * out_strides[2] + ow * out_strides[3];
            tmp += grad_out[out_i];
        }
    }

    grad_inp[i] += tmp / static_cast<float>(op.kernel * op.kernel);
}

extern "C" __global__ void max_pool2d_forward(
    const Pool2dOp op,
    const size_t *inp_strides,
    const size_t *out_strides,
    const float *inp, // 4d (Batch, Channels, Height, Width)
    float *out // 4d (Batch, Channels, HeightOut, WidthOut)
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t numel = op.batch * op.chan * op.h_out * op.w_out;
    if (i >= numel) {
        return;
    }

    unsigned int idx = i;
    const size_t ow = idx % op.w_out;
    idx /= op.w_out;
    const size_t oh = idx % op.h_out;
    idx /= op.h_out;
    const size_t c = idx % op.chan;
    idx /= op.chan;
    const size_t b = idx % op.batch;
    idx /= op.batch;

    float tmp = -INFINITY;
    for(size_t k1 = 0; k1 < op.kernel; k1++) {
        for (size_t k2 = 0; k2 < op.kernel; k2++) {
            const size_t y_plus_p = oh * op.stride + k1;
            if (y_plus_p < op.padding) { continue; }
            const size_t y = y_plus_p - op.padding;
            if (y >= op.h_in) { continue; }
            const size_t x_plus_p = ow * op.stride + k2;
            if (x_plus_p < op.padding) { continue; }
            const size_t x = x_plus_p - op.padding;
            if (x >= op.w_in) { continue; }

            auto inp_i = b * inp_strides[0] + c * inp_strides[1] + y * inp_strides[2] + x * inp_strides[3];
            tmp = fmaxf(tmp, inp[inp_i]);
        }
    }

    out[i] = tmp;
}

extern "C" __global__ void max_pool2d_backward(
    const Pool2dOp op,
    const size_t *inp_strides,
    const size_t *out_strides,
    const float *inp, // 4d (Batch, Channels, Height, Width)
    float *grad_inp,
    const float *out, // 4d (Batch, Channels, HeightOut, WidthOut)
    const float *grad_out
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t numel = op.batch * op.chan * op.h_in * op.w_in;
    if (i >= numel) {
        return;
    }

    unsigned int idx = i;
    const size_t x = idx % op.w_in;
    idx /= op.w_in;
    const size_t y = idx % op.h_in;
    idx /= op.h_in;
    const size_t c = idx % op.chan;
    idx /= op.chan;
    const size_t b = idx % op.batch;
    idx /= op.batch;

    const float inp_v = inp[i];

    float tmp = 0.0;
    for(size_t k1 = 0; k1 < op.kernel; k1++) {
        for (size_t k2 = 0; k2 < op.kernel; k2++) {
            size_t oh = y + op.padding;
            if (oh < k1) { continue; }
            oh -= k1;
            if (oh % op.stride != 0) { continue; }
            oh /= op.stride;
            if (oh >= op.h_out) { continue; }

            size_t ow = x + op.padding;
            if (ow < k2) { continue; }
            ow -= k2;
            if (ow % op.stride != 0) { continue; }
            ow /= op.stride;
            if (ow >= op.w_out) { continue; }

            auto out_i = b * out_strides[0] + c * out_strides[1] + oh * out_strides[2] + ow * out_strides[3];

            if (out[out_i] == inp_v) {
                tmp += grad_out[out_i];
            }
        }
    }

    grad_inp[i] += tmp;
}


extern "C" __global__ void min_pool2d_forward(
    const Pool2dOp op,
    const size_t *inp_strides,
    const size_t *out_strides,
    const float *inp, // 4d (Batch, Channels, Height, Width)
    float *out // 4d (Batch, Channels, HeightOut, WidthOut)
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t numel = op.batch * op.chan * op.h_out * op.w_out;
    if (i >= numel) {
        return;
    }

    unsigned int idx = i;
    const size_t ow = idx % op.w_out;
    idx /= op.w_out;
    const size_t oh = idx % op.h_out;
    idx /= op.h_out;
    const size_t c = idx % op.chan;
    idx /= op.chan;
    const size_t b = idx % op.batch;
    idx /= op.batch;

    float tmp = INFINITY;
    for(size_t k1 = 0; k1 < op.kernel; k1++) {
        for (size_t k2 = 0; k2 < op.kernel; k2++) {
            const size_t y_plus_p = oh * op.stride + k1;
            if (y_plus_p < op.padding) { continue; }
            const size_t y = y_plus_p - op.padding;
            if (y >= op.h_in) { continue; }
            const size_t x_plus_p = ow * op.stride + k2;
            if (x_plus_p < op.padding) { continue; }
            const size_t x = x_plus_p - op.padding;
            if (x >= op.w_in) { continue; }

            auto inp_i = b * inp_strides[0] + c * inp_strides[1] + y * inp_strides[2] + x * inp_strides[3];
            tmp = fminf(tmp, inp[inp_i]);
        }
    }

    out[i] = tmp;
}

extern "C" __global__ void min_pool2d_backward(
    const Pool2dOp op,
    const size_t *inp_strides,
    const size_t *out_strides,
    const float *inp, // 4d (Batch, Channels, Height, Width)
    float *grad_inp,
    const float *out, // 4d (Batch, Channels, HeightOut, WidthOut)
    const float *grad_out
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t numel = op.batch * op.chan * op.h_in * op.w_in;
    if (i >= numel) {
        return;
    }

    unsigned int idx = i;
    const size_t x = idx % op.w_in;
    idx /= op.w_in;
    const size_t y = idx % op.h_in;
    idx /= op.h_in;
    const size_t c = idx % op.chan;
    idx /= op.chan;
    const size_t b = idx % op.batch;
    idx /= op.batch;

    const float inp_v = inp[i];

    float tmp = 0.0;
    for(size_t k1 = 0; k1 < op.kernel; k1++) {
        for (size_t k2 = 0; k2 < op.kernel; k2++) {
            size_t oh = y + op.padding;
            if (oh < k1) { continue; }
            oh -= k1;
            if (oh % op.stride != 0) { continue; }
            oh /= op.stride;
            if (oh >= op.h_out) { continue; }

            size_t ow = x + op.padding;
            if (ow < k2) { continue; }
            ow -= k2;
            if (ow % op.stride != 0) { continue; }
            ow /= op.stride;
            if (ow >= op.w_out) { continue; }

            auto out_i = b * out_strides[0] + c * out_strides[1] + oh * out_strides[2] + ow * out_strides[3];

            if (out[out_i] == inp_v) {
                tmp += grad_out[out_i];
            }
        }
    }

    grad_inp[i] += tmp;
}
