#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"

struct BinaryMulKernalOp {};

extern "C" __global__ void binary_mul_forward(
    const BinaryMulKernalOp op,
    const size_t numel,
    const size_t num_dims,
    const size_t *dims,
    const float *lhs,
    const size_t *lhs_strides,
    const float *rhs,
    const size_t *rhs_strides,
    float *out,
    const size_t *out_strides
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    unsigned int lhs_i = get_strided_index(i, num_dims, dims, lhs_strides);
    unsigned int rhs_i = get_strided_index(i, num_dims, dims, rhs_strides);
    unsigned int out_i = get_strided_index(i, num_dims, dims, out_strides);

    out[out_i] = lhs[lhs_i] * rhs[rhs_i];
}

extern "C" __global__ void binary_mul_backward(
    const BinaryMulKernalOp op,
    const size_t numel,
    const size_t num_dims,
    const size_t *dims,
    const float *lhs,
    float *grad_lhs,
    const size_t *lhs_strides,
    const float *rhs,
    float *grad_rhs,
    const size_t *rhs_strides,
    const float *grad_out,
    const size_t *out_strides
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    unsigned int lhs_i = get_strided_index(i, num_dims, dims, lhs_strides);
    unsigned int rhs_i = get_strided_index(i, num_dims, dims, rhs_strides);
    unsigned int out_i = get_strided_index(i, num_dims, dims, out_strides);

    auto x = lhs[lhs_i];
    auto y = rhs[rhs_i];
    auto go = grad_out[out_i];

    atomicAdd(grad_lhs + lhs_i, y * go);
    atomicAdd(grad_rhs + rhs_i, x * go);
}
