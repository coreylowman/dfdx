#include "hip/hip_runtime.h"
#include "unary_op_macros.cuh"

#define SIGMOID_f32(X) (1.0 / (1.0 + expf(-X))) 
#define SIGMOID_f64(X) (1.0 / (1.0 + exp(-X))) 

struct SigmoidKernelOp {};

UNARY_OP(float, sigmoid_fwd_f32, sigmoid_bwd_f32, SigmoidKernelOp,
        SIGMOID_f32(x),
        SIGMOID_f32(x) * (1.0 - SIGMOID_f32(x)))

UNARY_OP(double, sigmoid_fwd_f64, sigmoid_bwd_f64, SigmoidKernelOp,
        SIGMOID_f64(x),
        SIGMOID_f64(x) * (1.0 - SIGMOID_f64(x)))
        