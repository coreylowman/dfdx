#include "hip/hip_runtime.h"
#include "unary_op_macros.cuh"

#define SIGMOID_f32(X) (1.0 / (1.0 + expf(-X))) 
#define SIGMOID_f64(X) (1.0 / (1.0 + exp(-X))) 

struct SigmoidKernelOp {};

UNARY_OP(float, sigmoid_fwd_f32, sigmoid_bwd_f32, SigmoidKernelOp,
        SIGMOID_f32(x),
        y * (1.0 - y))

UNARY_OP(double, sigmoid_fwd_f64, sigmoid_bwd_f64, SigmoidKernelOp,
        SIGMOID_f64(x),
        y * (1.0 - y))
        