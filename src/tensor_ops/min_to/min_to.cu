#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"

// atomicMax is not implemented for floats,
// solution copied https://stackoverflow.com/questions/17399119/how-do-i-use-atomicmax-on-floating-point-values-in-cuda
__device__ __forceinline__ float atomicMinf(float * addr, float value) {
    if (signbit(value)) {
        return __uint_as_float(atomicMax((unsigned int *)addr, __float_as_uint(value)));
    } else {
        return __int_as_float(atomicMin((int *)addr, __float_as_int(value)));
    }
}

__device__ __forceinline__ double atomicMinf(double * addr, double value) {
    if (signbit(value)) {
        return __longlong_as_double(atomicMax((unsigned long long int *)addr, __double_as_longlong(value)));
    } else {
        return __longlong_as_double(atomicMin((long long int *)addr, __double_as_longlong(value)));
    }
}

__device__ __forceinline__ float fminNonAtomic(float a, float b) {
    return fminf(a, b);
}

__device__ __forceinline__ double fminNonAtomic(double a, double b) {
    return fmin(a, b);
}

// Efficiently computes the min of each chunk in "data" of size chunk_len, and
// stores the minimums in out[i / chunk_len]
template<typename T>
__device__ void chunk_min(
    const size_t numel,
    const size_t chunk_len,
    const T data,
    T* out
) {
    __shared__ T buf[1024];
    // assumes that threads where i >= numel have already exited
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int block_i = threadIdx.x;
    buf[block_i] = data;

    unsigned int chunk_i = i % chunk_len;
    unsigned int chunk_start = max((int)(block_i - chunk_i), 0);
    unsigned int chunk_end = min((unsigned int)(block_i + chunk_len - chunk_i), blockDim.x);

    chunk_i = block_i - chunk_start;

    size_t max_chunk_len = min(chunk_end - chunk_start, blockDim.x);
    size_t incr = next_power_of_two(max_chunk_len) >> 1;

    __syncthreads();

    // Uses sequential addressing as discussed in
    // https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
    for (; incr > 0; incr >>= 1) {
        unsigned int block_i_2 = block_i + incr;

        if (block_i_2 < chunk_end && chunk_i < incr) {
            // This is sound because __syncthreads and the conditions above
            // ensure that no data races occur
            buf[block_i] = fminNonAtomic(buf[block_i], buf[block_i_2]);
        }

        __syncthreads();
    }

    if (block_i == chunk_start) {
        atomicMinf(out + i / chunk_len, buf[block_i]);
    }
}

// strides and dims specify how to index inp to put all summed elements next to
// each other, and chunk_len is len(inp) / len(out)
template<typename T>
__device__ void min_to_forward(
    const size_t numel,
    const size_t num_dims,
    const size_t chunk_len,
    const T *inp,
    const size_t *dims,
    const size_t *strides,
    T *out
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= numel) {
        return;
    }

    unsigned int inp_i = get_strided_index(i, num_dims, dims, strides);
    chunk_min(numel, chunk_len, inp[inp_i], out);
}

// Accepts pre-broadcasted strides for both input & output.
// So both inp & out are expected to be broadcasted to the same size.
template<typename T>
__device__ void min_to_backward(
    const size_t numel,
    const size_t num_dims,
    const T elems_per_thread,
    const size_t *dims,
    const T *inp,
    T *grad_inp,
    const size_t *inp_strides,
    const T *out,
    const T *grad_out,
    const size_t *out_strides
) {
    unsigned int inp_i = blockIdx.x * blockDim.x + threadIdx.x;

    if (inp_i >= numel) {
        return;
    }

    unsigned int i = get_unstrided_index(inp_i, num_dims, dims, inp_strides);
    unsigned int out_i = get_strided_index(i, num_dims, dims, out_strides);

    auto tmp = inp[inp_i] == out[out_i] ? grad_out[out_i] : 0.0;
    grad_inp[inp_i] += tmp * elems_per_thread;
}

extern "C" __global__ void min_to_forward_f32(
    const size_t numel,
    const size_t num_dims,
    const size_t chunk_len,
    const float *inp,
    const size_t *dims,
    const size_t *strides,
    float *out
) {
    min_to_forward(numel, num_dims, chunk_len, inp, dims, strides, out);
}

extern "C" __global__ void min_to_backward_f32(
    const size_t numel,
    const size_t num_dims,
    const float elems_per_thread,
    const size_t *dims,
    const float *inp,
    float *grad_inp,
    const size_t *inp_strides,
    const float *out,
    const float *grad_out,
    const size_t *out_strides
) {
    min_to_backward(numel, num_dims, elems_per_thread, dims, inp, grad_inp, inp_strides, out, grad_out, out_strides);
}

extern "C" __global__ void min_to_forward_f64(
    const size_t numel,
    const size_t num_dims,
    const size_t chunk_len,
    const double *inp,
    const size_t *dims,
    const size_t *strides,
    double *out
) {
    min_to_forward(numel, num_dims, chunk_len, inp, dims, strides, out);
}

extern "C" __global__ void min_to_backward_f64(
    const size_t numel,
    const size_t num_dims,
    const double elems_per_thread,
    const size_t *dims,
    const double *inp,
    double *grad_inp,
    const size_t *inp_strides,
    const double *out,
    const double *grad_out,
    const size_t *out_strides
) {
    min_to_backward(numel, num_dims, elems_per_thread, dims, inp, grad_inp, inp_strides, out, grad_out, out_strides);
}
