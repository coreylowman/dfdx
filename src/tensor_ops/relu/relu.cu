#include "hip/hip_runtime.h"
#include "unary_op_macros.cuh"

struct ReLUKernelOp {};

UNARY_OP(float, relu_fwd_f32, relu_bwd_f32, ReLUKernelOp,
        fmaxf(x, 0.0),
        x > 0.0 ? 1.0 : 0.0)

UNARY_OP(double, relu_fwd_f64, relu_bwd_f64, ReLUKernelOp,
        fmax(x, 0.0),
        x > 0.0 ? 1.0 : 0.0)
        