
#include <hip/hip_runtime.h>
struct BinaryMulKernalOp {};

extern "C" __global__ void binary_mul_forward(
    const BinaryMulKernalOp op,
    const size_t numel,
    const float *lhs,
    const float *rhs,
    float *out
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    out[i] = lhs[i] * rhs[i];
}

extern "C" __global__ void binary_mul_backward(
    const BinaryMulKernalOp op,
    const size_t numel,
    const float *lhs,
    float *grad_lhs,
    const float *rhs,
    float *grad_rhs,
    const float *grad_out
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    auto x = lhs[i];
    auto y = rhs[i];
    auto go = grad_out[i];

    grad_lhs[i] += y * go;
    grad_rhs[i] += x * go;
}
