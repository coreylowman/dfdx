#include "hip/hip_runtime.h"
#include "unary_op_macros.cuh"
#define _USE_MATH_DEFINES
#include <math.h>

struct AccurateGeLUKernelOp {};

template <typename T> __device__ T accurate_gelu_fwd(T x) {
    T one = 1.0;
    T half = 0.5;
    T alpha = M_SQRT1_2;
    return half * x * (one + erfg(x * alpha));
}

template <typename T> __device__ T accurate_gelu_bwd(T x) {
    T one = 1.0;
    T half = 0.5;
    T alpha = M_SQRT1_2;
    T scale = M_2_SQRTPI;
    T x_sq = x * x;
    T arg = -half * x_sq;
    T norm = scale * expg(arg);

    T left = half * x;
    T right = one + erfg(alpha * x);

    T left_derivative = half * right;

    T right_derivative = left * norm;

    return left_derivative + right_derivative;
}

UNARY_OP(__half, accurate_gelu_fwd_f16, accurate_gelu_bwd_f16,
         AccurateGeLUKernelOp, accurate_gelu_fwd(x), accurate_gelu_bwd(x))

UNARY_OP(float, accurate_gelu_fwd_f32, accurate_gelu_bwd_f32,
         AccurateGeLUKernelOp, accurate_gelu_fwd(x), accurate_gelu_bwd(x))

UNARY_OP(double, accurate_gelu_fwd_f64, accurate_gelu_bwd_f64,
         AccurateGeLUKernelOp, accurate_gelu_fwd(x), accurate_gelu_bwd(x))
