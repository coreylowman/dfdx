#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"

extern "C" __global__ void eq_forward(
    const size_t numel,
    const size_t num_dims,
    const size_t *dims,
    const float *lhs,
    const size_t *lhs_strides,
    const float *rhs,
    const size_t *rhs_strides,
    bool *out,
    const size_t *out_strides
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    unsigned int lhs_i = get_strided_index(i, num_dims, dims, lhs_strides);
    unsigned int rhs_i = get_strided_index(i, num_dims, dims, rhs_strides);
    unsigned int out_i = get_strided_index(i, num_dims, dims, out_strides);

    out[out_i] = lhs[lhs_i] == rhs[rhs_i];
}

extern "C" __global__ void scalar_eq_forward(
    const size_t numel,
    const size_t num_dims,
    const size_t *dims,
    const float *lhs,
    const size_t *lhs_strides,
    float scalar,
    bool *out,
    const size_t *out_strides
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    unsigned int lhs_i = get_strided_index(i, num_dims, dims, lhs_strides);
    unsigned int out_i = get_strided_index(i, num_dims, dims, out_strides);

    out[out_i] = lhs[lhs_i] == scalar;
}

extern "C" __global__ void ne_forward(
    const size_t numel,
    const size_t num_dims,
    const size_t *dims,
    const float *lhs,
    const size_t *lhs_strides,
    const float *rhs,
    const size_t *rhs_strides,
    bool *out,
    const size_t *out_strides
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    unsigned int lhs_i = get_strided_index(i, num_dims, dims, lhs_strides);
    unsigned int rhs_i = get_strided_index(i, num_dims, dims, rhs_strides);
    unsigned int out_i = get_strided_index(i, num_dims, dims, out_strides);

    out[out_i] = lhs[lhs_i] != rhs[rhs_i];
}

extern "C" __global__ void scalar_ne_forward(
    const size_t numel,
    const size_t num_dims,
    const size_t *dims,
    const float *lhs,
    const size_t *lhs_strides,
    float scalar,
    bool *out,
    const size_t *out_strides
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    unsigned int lhs_i = get_strided_index(i, num_dims, dims, lhs_strides);
    unsigned int out_i = get_strided_index(i, num_dims, dims, out_strides);

    out[out_i] = lhs[lhs_i] != scalar;
}

extern "C" __global__ void gt_forward(
    const size_t numel,
    const size_t num_dims,
    const size_t *dims,
    const float *lhs,
    const size_t *lhs_strides,
    const float *rhs,
    const size_t *rhs_strides,
    bool *out,
    const size_t *out_strides
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    unsigned int lhs_i = get_strided_index(i, num_dims, dims, lhs_strides);
    unsigned int rhs_i = get_strided_index(i, num_dims, dims, rhs_strides);
    unsigned int out_i = get_strided_index(i, num_dims, dims, out_strides);

    out[out_i] = lhs[lhs_i] > rhs[rhs_i];
}

extern "C" __global__ void scalar_gt_forward(
    const size_t numel,
    const size_t num_dims,
    const size_t *dims,
    const float *lhs,
    const size_t *lhs_strides,
    float scalar,
    bool *out,
    const size_t *out_strides
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    unsigned int lhs_i = get_strided_index(i, num_dims, dims, lhs_strides);
    unsigned int out_i = get_strided_index(i, num_dims, dims, out_strides);

    out[out_i] = lhs[lhs_i] > scalar;
}

extern "C" __global__ void ge_forward(
    const size_t numel,
    const size_t num_dims,
    const size_t *dims,
    const float *lhs,
    const size_t *lhs_strides,
    const float *rhs,
    const size_t *rhs_strides,
    bool *out,
    const size_t *out_strides
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    unsigned int lhs_i = get_strided_index(i, num_dims, dims, lhs_strides);
    unsigned int rhs_i = get_strided_index(i, num_dims, dims, rhs_strides);
    unsigned int out_i = get_strided_index(i, num_dims, dims, out_strides);

    out[out_i] = lhs[lhs_i] >= rhs[rhs_i];
}

extern "C" __global__ void scalar_ge_forward(
    const size_t numel,
    const size_t num_dims,
    const size_t *dims,
    const float *lhs,
    const size_t *lhs_strides,
    float scalar,
    bool *out,
    const size_t *out_strides
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    unsigned int lhs_i = get_strided_index(i, num_dims, dims, lhs_strides);
    unsigned int out_i = get_strided_index(i, num_dims, dims, out_strides);

    out[out_i] = lhs[lhs_i] >= scalar;
}

extern "C" __global__ void lt_forward(
    const size_t numel,
    const size_t num_dims,
    const size_t *dims,
    const float *lhs,
    const size_t *lhs_strides,
    const float *rhs,
    const size_t *rhs_strides,
    bool *out,
    const size_t *out_strides
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    unsigned int lhs_i = get_strided_index(i, num_dims, dims, lhs_strides);
    unsigned int rhs_i = get_strided_index(i, num_dims, dims, rhs_strides);
    unsigned int out_i = get_strided_index(i, num_dims, dims, out_strides);

    out[out_i] = lhs[lhs_i] < rhs[rhs_i];
}

extern "C" __global__ void scalar_lt_forward(
    const size_t numel,
    const size_t num_dims,
    const size_t *dims,
    const float *lhs,
    const size_t *lhs_strides,
    float scalar,
    bool *out,
    const size_t *out_strides
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    unsigned int lhs_i = get_strided_index(i, num_dims, dims, lhs_strides);
    unsigned int out_i = get_strided_index(i, num_dims, dims, out_strides);

    out[out_i] = lhs[lhs_i] < scalar;
}

extern "C" __global__ void le_forward(
    const size_t numel,
    const size_t num_dims,
    const size_t *dims,
    const float *lhs,
    const size_t *lhs_strides,
    const float *rhs,
    const size_t *rhs_strides,
    bool *out,
    const size_t *out_strides
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    unsigned int lhs_i = get_strided_index(i, num_dims, dims, lhs_strides);
    unsigned int rhs_i = get_strided_index(i, num_dims, dims, rhs_strides);
    unsigned int out_i = get_strided_index(i, num_dims, dims, out_strides);

    out[out_i] = lhs[lhs_i] <= rhs[rhs_i];
}

extern "C" __global__ void scalar_le_forward(
    const size_t numel,
    const size_t num_dims,
    const size_t *dims,
    const float *lhs,
    const size_t *lhs_strides,
    float scalar,
    bool *out,
    const size_t *out_strides
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    unsigned int lhs_i = get_strided_index(i, num_dims, dims, lhs_strides);
    unsigned int out_i = get_strided_index(i, num_dims, dims, out_strides);

    out[out_i] = lhs[lhs_i] <= scalar;
}
