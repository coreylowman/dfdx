#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"

extern "C" __global__ void eq_forward(
    const size_t numel,
    const size_t num_dims,
    const size_t *dims,
    const float *lhs,
    const size_t *lhs_strides,
    const float *rhs,
    const size_t *rhs_strides,
    bool *out,
    const size_t *out_strides
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    unsigned int lhs_i = get_strided_index(i, num_dims, dims, lhs_strides);
    unsigned int rhs_i = get_strided_index(i, num_dims, dims, rhs_strides);
    unsigned int out_i = get_strided_index(i, num_dims, dims, out_strides);

    out[out_i] = lhs[lhs_i] == rhs[rhs_i];
}
