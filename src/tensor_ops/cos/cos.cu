#include "hip/hip_runtime.h"
#include "unary_op_macros.cuh"

struct CosKernelOp {};

UNARY_OP(float, cos_fwd_f32, cos_bwd_f32, CosKernelOp,
        cosf(x),
        -sinf(x))

UNARY_OP(double, cos_fwd_f64, cos_bwd_f64, CosKernelOp,
        cos(x),
        -sin(x))
        