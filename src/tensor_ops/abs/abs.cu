#include "hip/hip_runtime.h"
#include "unary_op_macros.cuh"

struct AbsKernelOp {};

UNARY_OP(float, abs_fwd_f32, abs_bwd_f32, AbsKernelOp,
        fabsf(x),
        x == 0.0 ? 0.0 : copysignf(1.0, x));

UNARY_OP(double, abs_fwd_f64, abs_bwd_f64, AbsKernelOp,
        fabs(x),
        x == 0.0 ? 0.0 : copysign(1.0, x));
