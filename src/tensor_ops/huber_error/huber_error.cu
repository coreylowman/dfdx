#include "hip/hip_runtime.h"
#include "binary_op_macros.cuh"

struct HuberErrorOp {
    float delta;
};

LONG_BINARY_OP(huber_error_forward, huber_error_backward, HuberErrorOp,
    {
        float a = x - y;

        if (fabsf(a) < op.delta) {
            fx = a * a * 0.5;
        } else {
            fx = op.delta * (fabsf(a) - 0.5 * op.delta);
        }
    },
    {
        auto a = x - y;

        if (a == 0.0) {
            dfdx = 0.0;
        } else if (fabsf(a) < op.delta) {
            dfdx = a;
        } else {
            dfdx = copysignf(op.delta, a);
        }

        dfdy = -dfdx;
    }
)
