
#include <hip/hip_runtime.h>
struct HuberErrorOp {
    float delta;
};

__device__ unsigned int get_strided_index(
    unsigned int idx,
    const size_t num_dims,
    const size_t *dims,
    const size_t *strides
) {
    unsigned int strided_i = 0;
    for (unsigned int d = 0; d < num_dims; d++) {
        unsigned int dim_idx = num_dims - 1 - d;
        strided_i += (idx % dims[dim_idx]) * strides[dim_idx];
        idx /= dims[dim_idx];
    }
    return strided_i;
}

extern "C" __global__ void huber_error_forward(
    const HuberErrorOp op,
    const size_t numel,
    const size_t num_dims,
    const size_t *dims,
    const float *lhs,
    const size_t *lhs_strides,
    const float *rhs,
    const size_t *rhs_strides,
    float *out,
    const size_t *out_strides
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    unsigned int lhs_i = get_strided_index(i, num_dims, dims, lhs_strides);
    unsigned int rhs_i = get_strided_index(i, num_dims, dims, rhs_strides);
    unsigned int out_i = get_strided_index(i, num_dims, dims, out_strides);

    float a = lhs[lhs_i] - rhs[rhs_i];

    if (fabsf(a) < op.delta) {
        out[out_i] = a * a * 0.5;
    } else {
        out[out_i] = op.delta * (fabsf(a) - 0.5 * op.delta);
    }
}

extern "C" __global__ void huber_error_backward(
    const HuberErrorOp op,
    const size_t numel,
    const size_t num_dims,
    const size_t *dims,
    const float *lhs,
    float *grad_lhs,
    const size_t *lhs_strides,
    const float *rhs,
    float *grad_rhs,
    const size_t *rhs_strides,
    const float *grad_out,
    const size_t *out_strides
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    unsigned int lhs_i = get_strided_index(i, num_dims, dims, lhs_strides);
    unsigned int rhs_i = get_strided_index(i, num_dims, dims, rhs_strides);
    unsigned int out_i = get_strided_index(i, num_dims, dims, out_strides);

    auto a = lhs[lhs_i] - rhs[rhs_i];
    auto go = grad_out[out_i];

    float dfdx, dfdy;

    if (a == 0.0) {
        dfdx = 0.0;
    } else if (fabsf(a) < op.delta) {
        dfdx = a;
    } else {
        dfdx = copysignf(op.delta, a);
    }

    dfdy = -dfdx;

    grad_lhs[lhs_i] += dfdx * go;
    grad_rhs[rhs_i] += dfdy * go;
}
