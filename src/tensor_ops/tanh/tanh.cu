#include "hip/hip_runtime.h"
#include "unary_op_macros.cuh"

struct TanhKernelOp {};

UNARY_OP(float, tanh_fwd_f32, tanh_bwd_f32, TanhKernelOp,
        tanhf(x),
        1 - y * y)

UNARY_OP(double, tanh_fwd_f64, tanh_bwd_f64, TanhKernelOp,
        tanh(x),
        1 - y * y)
        