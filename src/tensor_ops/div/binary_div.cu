
#include <hip/hip_runtime.h>
__device__ unsigned int get_strided_index(
    unsigned int idx,
    const size_t num_dims,
    const size_t *dims,
    const size_t *strides
) {
    unsigned int strided_i = 0;
    for (unsigned int d = 0; d < num_dims; d++) {
        unsigned int dim_idx = num_dims - 1 - d;
        strided_i += (idx % dims[dim_idx]) * strides[dim_idx];
        idx /= dims[dim_idx];
    }
    return strided_i;
}

extern "C" __global__ void binary_div_forward(
    const size_t numel,
    const size_t num_dims,
    const size_t *dims,
    const float *lhs,
    const size_t *lhs_strides,
    const float *rhs,
    const size_t *rhs_strides,
    float *out,
    const size_t *out_strides
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    unsigned int lhs_i = get_strided_index(i, num_dims, dims, lhs_strides);
    unsigned int rhs_i = get_strided_index(i, num_dims, dims, rhs_strides);
    unsigned int out_i = get_strided_index(i, num_dims, dims, out_strides);

    out[out_i] = lhs[lhs_i] / rhs[rhs_i];
}

extern "C" __global__ void binary_div_backward(
    const size_t numel,
    const size_t num_dims,
    const size_t *dims,
    const float *lhs,
    float *grad_lhs,
    const size_t *lhs_strides,
    const float *rhs,
    float *grad_rhs,
    const size_t *rhs_strides,
    const float *grad_out,
    const size_t *out_strides
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numel) {
        return;
    }

    unsigned int lhs_i = get_strided_index(i, num_dims, dims, lhs_strides);
    unsigned int rhs_i = get_strided_index(i, num_dims, dims, rhs_strides);
    unsigned int out_i = get_strided_index(i, num_dims, dims, out_strides);

    auto x = rhs[rhs_i];
    auto y = lhs[lhs_i];
    auto go = grad_out[out_i];

    float dfdx = 1.0 / y;
    grad_lhs[lhs_i] += dfdx * go;

    float dfdy = -x / (y * y);
    grad_rhs[rhs_i] += dfdy * go;
}
