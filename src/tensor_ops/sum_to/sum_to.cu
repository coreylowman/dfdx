#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"

// Efficiently computes the sum of each chunk in "data" of size chunk_len, and
// stores the sums in out[i / chunk_len]
template<typename T>
__device__ void chunk_sum(
    const size_t numel,
    const size_t chunk_len,
    const T data,
    T* out
) {
    __shared__ T buf[1024];
    // assumes that threads where i >= numel have already exited
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int block_i = threadIdx.x;
    buf[block_i] = data;

    unsigned int chunk_i = i % chunk_len;
    unsigned int chunk_start = max((int)(block_i - chunk_i), 0);
    unsigned int chunk_end = min((unsigned int)(block_i + chunk_len - chunk_i), blockDim.x);

    chunk_i = block_i - chunk_start;

    size_t max_chunk_len = min(chunk_end - chunk_start, blockDim.x);
    size_t incr = next_power_of_two(max_chunk_len) >> 1;

    __syncthreads();

    // Uses sequential addressing as discussed in
    // https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
    for (; incr > 0; incr >>= 1) {
        unsigned int block_i_2 = block_i + incr;

        if (block_i_2 < chunk_end && chunk_i < incr) {
            // This is sound because __syncthreads and the conditions above
            // ensure that no data races occur
            buf[block_i] += buf[block_i_2];
        }

        __syncthreads();
    }

    if (block_i == chunk_start) {
        atomicAdd(out + i / chunk_len, buf[block_i]);
    }
}

// strides and dims specify how to index inp to put all summed elements next to
// each other, and chunk_len is len(inp) / len(out)
template<typename T>
__device__ void sum_to_fwd(
    const size_t numel,
    const size_t num_dims,
    const T elems_per_thread,
    const size_t chunk_len,
    const T *inp,
    const size_t *dims,
    const size_t *strides,
    T *out
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= numel) {
        return;
    }

    unsigned int inp_i = get_strided_index(i, num_dims, dims, strides);
    chunk_sum(numel, chunk_len, inp[inp_i] * elems_per_thread, out);
}

// Accepts pre-broadcasted strides for both input & output.
// So both inp & out are expected to be broadcasted to the same size.
template<typename T>
__device__ void sum_to_bwd(
    const size_t numel,
    const size_t num_dims,
    const T elems_per_thread,
    const size_t *dims,
    T *grad_inp,
    const size_t *inp_strides,
    const T *grad_out,
    const size_t *out_strides
) {
    unsigned int inp_i = blockIdx.x * blockDim.x + threadIdx.x;

    if (inp_i >= numel) {
        return;
    }

    unsigned int i = get_unstrided_index(inp_i, num_dims, dims, inp_strides);
    unsigned int out_i = get_strided_index(i, num_dims, dims, out_strides);
    auto tmp = grad_out[out_i];

    // NOTE: since size of output is less than input, only 1 thread will be writing to inp
    // at a time. this means we don't have to worry about multiple concurrent writes
    // like we do with fwd.
    grad_inp[inp_i] += tmp * elems_per_thread;
}

#define SUM(TYPENAME, FWD, BWD) \
extern "C" __global__ void FWD( \
    const size_t numel, \
    const size_t num_dims, \
    const TYPENAME elems_per_thread, \
    const size_t chunk_len, \
    const TYPENAME *inp, \
    const size_t *dims, \
    const size_t *strides, \
    TYPENAME *out \
) { \
    sum_to_fwd(numel, num_dims, elems_per_thread, chunk_len, inp, dims, strides, out); \
} \
extern "C" __global__ void BWD( \
    const size_t numel, \
    const size_t num_dims, \
    const TYPENAME elems_per_thread, \
    const size_t *dims, \
    TYPENAME *grad_inp, \
    const size_t *inp_strides, \
    const TYPENAME *grad_out, \
    const size_t *out_strides \
) { \
    sum_to_bwd(numel, num_dims, elems_per_thread, dims, grad_inp, inp_strides, grad_out, out_strides); \
}

SUM(float, sum_to_fwd_f32, sum_to_bwd_f32);
SUM(double, sum_to_fwd_f64, sum_to_bwd_f64);
