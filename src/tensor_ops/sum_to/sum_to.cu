
#include <hip/hip_runtime.h>
__device__ unsigned int get_strided_index(
    unsigned int idx,
    size_t num_dims,
    const size_t *dims,
    const size_t *strides
) {
    unsigned int strided_i = 0;
    for (unsigned int d = 0; d < num_dims; d++) {
        unsigned int dim_idx = num_dims - 1 - d;
        strided_i += (idx % dims[dim_idx]) * strides[dim_idx];
        idx /= dims[dim_idx];
    }
    return strided_i;
}

__device__ unsigned int get_unstrided_index(
    const unsigned int strided_i,
    const size_t num_dims,
    const size_t *dims,
    const size_t *strides
) {
    unsigned int idx = 0;
    for (unsigned int d = 0; d < num_dims; d++) {
        idx *= dims[d];
        idx += strides[d] == 0 ? 0 : (strided_i / strides[d]) % dims[d];
    }
    return idx;
}

// Accepts pre-broadcasted strides for both input & output.
// So both inp & out are expected to be broadcasted to the same size.
extern "C" __global__ void sum_to_forward(
    const size_t numel,
    const size_t num_dims,
    const float mul,
    const size_t *dims,
    const float *inp,
    const size_t *inp_strides,
    float *out,
    const size_t *out_strides
) {
    unsigned int inp_i = blockIdx.x * blockDim.x + threadIdx.x;

    if (inp_i >= numel) {
        return;
    }

    auto tmp = inp[inp_i];

    unsigned int i = get_unstrided_index(inp_i, num_dims, dims, inp_strides);
    unsigned int out_i = get_strided_index(i, num_dims, dims, out_strides);
    atomicAdd(out + out_i, tmp * mul);
}

// Accepts pre-broadcasted strides for both input & output.
// So both inp & out are expected to be broadcasted to the same size.
extern "C" __global__ void sum_to_backward(
    const size_t numel,
    const size_t num_dims,
    const float mul,
    const size_t *dims,
    float *grad_inp,
    const size_t *inp_strides,
    const float *grad_out,
    const size_t *out_strides
) {
    unsigned int inp_i = blockIdx.x * blockDim.x + threadIdx.x;

    if (inp_i >= numel) {
        return;
    }

    unsigned int i = get_unstrided_index(inp_i, num_dims, dims, inp_strides);
    unsigned int out_i = get_strided_index(i, num_dims, dims, out_strides);
    auto tmp = grad_out[out_i];

    // NOTE: since size of output is less than input, only 1 thread will be writing to inp
    // at a time. this means we don't have to worry about multiple concurrent writes
    // like we do with forward.
    grad_inp[inp_i] += tmp * mul;
}
