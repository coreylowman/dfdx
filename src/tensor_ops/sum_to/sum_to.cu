#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"

// Efficiently computes the sum of each chunk in "data" of size chunk_len, and
// stores the sums in out[i / chunk_len]
__device__ void chunk_sum(
    const size_t numel,
    const size_t chunk_len,
    const float data,
    float* out
) {
    __shared__ float buf[1024];
    // assumes that threads where i >= numel have already exited
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int block_i = threadIdx.x;
    buf[block_i] = data;

    unsigned int chunk_i = i % chunk_len;
    unsigned int chunk_start = max((int)(block_i - chunk_i), 0);
    unsigned int chunk_end = min((unsigned int)(block_i + chunk_len - chunk_i), blockDim.x);

    chunk_i = block_i - chunk_start;

    size_t max_chunk_len = min(chunk_end - chunk_start, blockDim.x);
    size_t incr = next_power_of_two(max_chunk_len) >> 1;

    __syncthreads();

    // Uses sequential addressing as discussed in
    // https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
    for (; incr > 0; incr >>= 1) {
        unsigned int block_i_2 = block_i + incr;

        if (block_i_2 < chunk_end && chunk_i < incr) {
            // This is sound because __syncthreads and the conditions above
            // ensure that no data races occur
            buf[block_i] += buf[block_i_2];
        }

        __syncthreads();
    }

    if (block_i == chunk_start) {
        atomicAdd(out + i / chunk_len, buf[block_i]);
    }
}

// strides and dims specify how to index inp to put all summed elements next to
// each other, and chunk_len is len(inp) / len(out)
extern "C" __global__ void sum_to_forward(
    const size_t numel,
    const size_t num_dims,
    const float elems_per_thread,
    const size_t chunk_len,
    const float *inp,
    const size_t *dims,
    const size_t *strides,
    float *out
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= numel) {
        return;
    }

    unsigned int inp_i = get_strided_index(i, num_dims, dims, strides);
    chunk_sum(numel, chunk_len, inp[inp_i] * elems_per_thread, out);
}

// Accepts pre-broadcasted strides for both input & output.
// So both inp & out are expected to be broadcasted to the same size.
extern "C" __global__ void sum_to_backward(
    const size_t numel,
    const size_t num_dims,
    const float elems_per_thread,
    const size_t *dims,
    float *grad_inp,
    const size_t *inp_strides,
    const float *grad_out,
    const size_t *out_strides
) {
    unsigned int inp_i = blockIdx.x * blockDim.x + threadIdx.x;

    if (inp_i >= numel) {
        return;
    }

    unsigned int i = get_unstrided_index(inp_i, num_dims, dims, inp_strides);
    unsigned int out_i = get_strided_index(i, num_dims, dims, out_strides);
    auto tmp = grad_out[out_i];

    // NOTE: since size of output is less than input, only 1 thread will be writing to inp
    // at a time. this means we don't have to worry about multiple concurrent writes
    // like we do with forward.
    grad_inp[inp_i] += tmp * elems_per_thread;
}
