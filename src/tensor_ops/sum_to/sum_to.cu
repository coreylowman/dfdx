
#include <hip/hip_runtime.h>
__device__ unsigned int get_strided_index(
    unsigned int idx,
    size_t num_dims,
    const size_t *dims,
    const size_t *strides
) {
    unsigned int strided_i = 0;
    for (unsigned int d = 0; d < num_dims; d++) {
        unsigned int dim_idx = num_dims - 1 - d;
        strided_i += (idx % dims[dim_idx]) * strides[dim_idx];
        idx /= dims[dim_idx];
    }
    return strided_i;
}

__device__ unsigned int get_unstrided_index(
    const unsigned int strided_i,
    const size_t num_dims,
    const size_t *dims,
    const size_t *strides
) {
    unsigned int idx = 0;
    for (unsigned int d = 0; d < num_dims; d++) {
        idx *= dims[d];
        idx += strides[d] == 0 ? 0 : (strided_i / strides[d]) % dims[d];
    }
    return idx;
}

__device__ __forceinline__ unsigned int next_power_of_two(unsigned int v) {
    // Sourced from https://graphics.stanford.edu/~seander/bithacks.html#RoundUpPowerOf2
    v--;
    v |= v >> 1;
    v |= v >> 2;
    v |= v >> 4;
    v |= v >> 8;
    v |= v >> 16;
    v++;
    return v;
}

// Efficiently computes the sum of each chunk in "data" of size chunk_len, and
// stores the sums in out[i / chunk_len]
__device__ void chunk_sum(
    const size_t numel,
    const size_t chunk_len,
    const float data,
    float* out
) {
    __shared__ float buf[1024];
    // assumes that threads where i >= numel have already exited
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int block_i = threadIdx.x;
    buf[block_i] = data;

    unsigned int chunk_i = i % chunk_len;
    unsigned int chunk_start = max((int)(block_i - chunk_i), 0);
    unsigned int chunk_end = min((unsigned int)(block_i + chunk_len - chunk_i), blockDim.x);

    chunk_i = block_i - chunk_start;

    size_t max_chunk_len = min(chunk_end - chunk_start, blockDim.x);
    size_t incr = next_power_of_two(max_chunk_len) >> 1;

    __syncthreads();

    // Uses sequential addressing as discussed in
    // https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
    for (; incr > 0; incr >>= 1) {
        unsigned int block_i_2 = block_i + incr;

        if (block_i_2 < chunk_end && chunk_i < incr) {
            // This is sound because __syncthreads and the conditions above
            // ensure that no data races occur
            buf[block_i] += buf[block_i_2];
        }

        __syncthreads();
    }

    if (block_i == chunk_start) {
        atomicAdd(out + i / chunk_len, buf[block_i]);
    }
}

// inp_strides and dims must have broadcasted dimensions removed
extern "C" __global__ void sum_to_forward(
    const size_t numel,
    const size_t num_dims,
    const float elems_per_thread,
    const size_t chunk_len,
    const size_t *dims,
    const float *inp,
    const size_t *inp_strides,
    float *out
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= numel) {
        return;
    }

    unsigned int inp_i = get_strided_index(i, num_dims, dims, inp_strides);
    chunk_sum(numel, chunk_len, inp[inp_i], out);
}

// Accepts pre-broadcasted strides for both input & output.
// So both inp & out are expected to be broadcasted to the same size.
extern "C" __global__ void sum_to_backward(
    const size_t numel,
    const size_t num_dims,
    const float elems_per_thread,
    const size_t *dims,
    float *grad_inp,
    const size_t *inp_strides,
    const float *grad_out,
    const size_t *out_strides
) {
    unsigned int inp_i = blockIdx.x * blockDim.x + threadIdx.x;

    if (inp_i >= numel) {
        return;
    }

    unsigned int i = get_unstrided_index(inp_i, num_dims, dims, inp_strides);
    unsigned int out_i = get_strided_index(i, num_dims, dims, out_strides);
    auto tmp = grad_out[out_i];

    // NOTE: since size of output is less than input, only 1 thread will be writing to inp
    // at a time. this means we don't have to worry about multiple concurrent writes
    // like we do with forward.
    grad_inp[inp_i] += tmp * elems_per_thread;
}
