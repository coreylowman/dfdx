#include "hip/hip_runtime.h"
#include "binary_op_macros.cuh"

struct BinaryMulKernalOp {};

BINARY_OP(__half, bmul_fwd_f16, bmul_bwd_lhs_f16, bmul_bwd_rhs_f16, BinaryMulKernalOp,
    x * y,
    y,
    x)

BINARY_OP(float, bmul_fwd_f32, bmul_bwd_lhs_f32, bmul_bwd_rhs_f32, BinaryMulKernalOp,
    x * y,
    y,
    x)

BINARY_OP(double, bmul_fwd_f64, bmul_bwd_lhs_f64, bmul_bwd_rhs_f64, BinaryMulKernalOp,
    x * y,
    y,
    x)
   
